#include "hip/hip_runtime.h"
#include "CosmicConstants.cuh"

#include <stdio.h>

__global__ void mykernel() {
	printf("value is %f\n", K0);
}

void setDT(float newDT) {
	hipMemcpyToSymbol(HIP_SYMBOL(dt), &newDT, sizeof(newDT));
}

void setK0(float newK0) {
	mykernel << <1, 1 >> >();
	hipMemcpyToSymbol(HIP_SYMBOL(K0), &newK0, sizeof(newK0), 0, hipMemcpyHostToDevice);
	mykernel << <1, 1 >> >();
}

