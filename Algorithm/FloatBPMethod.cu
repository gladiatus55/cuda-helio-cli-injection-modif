#include "FloatBPMethod.cuh"
#include "cuda_implementation/cudaErrorCheck.cuh"
#include "BPResult.cuh"

void FloatBPMethod::runAlgorithm(ParamsCarrier *singleTone){
	simulationInputBP simulation; 
	setThreadBlockSize(); 
	hiprandState_t *state;
	double *w;
	float *Tkininj, *pinj;
	trajectoryHistoryBP *history, *local_history;
	gpuErrchk( hipMallocManaged(&w, ((blockSize * threadSize) *sizeof(double))) );
	gpuErrchk( hipMallocManaged(&Tkininj, ((blockSize * threadSize) *sizeof(float))) );
	gpuErrchk( hipMallocManaged(&pinj, ((blockSize * threadSize) *sizeof(float))) );
	gpuErrchk( hipMallocManaged(&state, ((blockSize * threadSize) * sizeof(hiprandState_t))) );
	gpuErrchk( hipHostMalloc(&local_history, ((blockSize * threadSize) *sizeof(trajectoryHistoryBP))) );
	gpuErrchk( hipMalloc(&history, ((blockSize * threadSize) *sizeof(trajectoryHistoryBP))) ); 
	simulation.singleTone = singleTone; 
	simulation.history = history; 
	simulation.pinj = pinj; 
	simulation.local_history = local_history; 
	simulation.Tkininj= Tkininj; 
	simulation.state = state; 
	simulation.w = w; 
	simulation.threadSize = threadSize; 
	simulation.blockSize = blockSize; 
	runBPMethod(&simulation);
	gpuErrchk( hipFree(w) );
	gpuErrchk( hipFree(Tkininj) );
	gpuErrchk( hipFree(pinj) );
	gpuErrchk( hipFree(state) );
	gpuErrchk( hipFree(history) );
	gpuErrchk( hipHostFree(local_history) );
	AbstractAlgorithm *result;
	result = new BPResult(); 
	result->runAlgorithm(singleTone);
}

// Compute capability actual device 
void FloatBPMethod::setThreadBlockSize() {
	hipDeviceProp_t gpuProperties; 
	gpuErrchk( hipGetDeviceProperties(&gpuProperties, 0) );
	int computeCapability = gpuProperties.major * 100 + gpuProperties.minor * 10; 
	switch (computeCapability) {
		case 610:	blockSize = 32768;
				threadSize = 512;
				break;
		case 750:	blockSize = 16384;
				threadSize = 1024;
				break;
		default:	blockSize = 64;
				threadSize = 64;
				break;
	}
}