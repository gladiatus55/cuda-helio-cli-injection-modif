#include "FloatFWMethod.cuh"
#include "FWMethodResult.cuh"
#include "cuda_implementation/cudaErrorCheck.cuh"

void FloatFWMethod::runAlgorithm(ParamsCarrier *singleTone){
	simulationInput simulation; 
	setThreadBlockSize(); 
	hiprandState_t *state;
	double *w;
	float *pinj;
	trajectoryHistory *history, *local_history;
	gpuErrchk( hipMallocManaged(&w, ((blockSize * threadSize) *sizeof(double))) );
	gpuErrchk( hipMallocManaged(&pinj, ((blockSize * threadSize) *sizeof(float))) );
	gpuErrchk( hipMallocManaged(&state, ((blockSize * threadSize) * sizeof(hiprandState_t))) );
	gpuErrchk(hipHostMalloc(&local_history, ((BLOCK_SIZE * THREAD_SIZE * 10) *sizeof(trajectoryHistory))));
	gpuErrchk(hipMalloc(&history, ((BLOCK_SIZE * THREAD_SIZE * 10) *sizeof(trajectoryHistory))));
	simulation.singleTone = singleTone;
	simulation.history = history; 
	simulation.local_history = local_history; 
	simulation.pinj = pinj; 
	simulation.state = state; 
	simulation.w = w; 
	simulation.threadSize = threadSize; 
	simulation.blockSize = blockSize; 
	runFWMethod(&simulation);
	gpuErrchk( hipFree(w) );
	gpuErrchk( hipFree(pinj) );
	gpuErrchk( hipFree(state) );
	gpuErrchk( hipFree(history) );
	gpuErrchk( hipHostFree(local_history) );
	AbstractAlgorithm *result;
	result = new FWMethodResult(); 
	result->runAlgorithm(singleTone);
}

// Compute capability actual device 
void FloatFWMethod::setThreadBlockSize() {
	hipDeviceProp_t gpuProperties; 
	gpuErrchk( hipGetDeviceProperties(&gpuProperties, 0) );
	int computeCapability = gpuProperties.major * 100 + gpuProperties.minor * 10; 
	switch (computeCapability) {
		case 610:	blockSize = 65536;
				threadSize = 512;
				break;
		case 750:	blockSize = 32768;
				threadSize = 1024;
				break;
		default:	blockSize = 64;
				threadSize = 64;
				break;
	}
}